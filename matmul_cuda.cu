
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

// CUDA Kernel for multiplying matrices
__global__
void matmul(int workingShapeX, int workingShapeY, int currentShapeY, int newShapeY, 
            int64_t *mat1, int64_t *mat2, int64_t *out) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int j = (blockIdx.y * blockDim.y) + threadIdx.y;
  if(i < workingShapeX && j < workingShapeY){
    int64_t sum = 0;
    for (int k = 0; k < currentShapeY; k++) {
      sum += mat1[(i *  currentShapeY) + k] * 
             mat2[(k *      newShapeY) + j];
    }
    out[(i * workingShapeY) + j] = sum;
  }
}

int main(int argc, char *argv[]) {
  FILE* input = fopen(argv[1], "rb");
  if (input != NULL) {
    int *currentShape = (int *)malloc(8);
    int *workingShape = (int *)malloc(8);
    int *    newShape = (int *)malloc(8);
    int64_t *currentMatrix = NULL, *workingMatrix = NULL;

    // Read the number of Rows and Columns in this Matrix
    while ((fread(newShape, sizeof(char), 8, input)) > 0) {
      // Allocate this matrix
      int64_t* newMatrix = (int64_t*)malloc(newShape[0] * newShape[1] * sizeof(int64_t));
      // Read the full contents of the matrix
      int matrixBytes = fread(newMatrix, sizeof(int64_t), newShape[0] * newShape[1], input);

      bool readyToMultiply = currentMatrix != NULL;
      if(readyToMultiply) {
        // Allocate the Working Matrix
        workingShape[0]   = currentShape[0];
        workingShape[1]   =     newShape[1];
        int workingLength = workingShape[0] * workingShape[1] * sizeof(int64_t);
        int currentLength = currentShape[0] * currentShape[1] * sizeof(int64_t);
        int     newLength =     newShape[0] *     newShape[1] * sizeof(int64_t);
        workingMatrix = (int64_t*)malloc(workingLength);

        // Multiply Current and New Matrices together
        if (workingLength > 10000) { // Choose GPU or CPU based on Matrix Size
          // Allocate Matrices on GPU
          int64_t *d_workingMatrix; hipMalloc(&d_workingMatrix, workingLength);
          int64_t *d_currentMatrix; hipMalloc(&d_currentMatrix, currentLength);
          int64_t *d_newMatrix    ; hipMalloc(&    d_newMatrix,     newLength);

          // Copy Matrices to GPU
          hipMemcpy(d_currentMatrix, currentMatrix, currentLength, hipMemcpyHostToDevice);
          hipMemcpy(    d_newMatrix,     newMatrix,     newLength, hipMemcpyHostToDevice);

          // Multiply Matrices on GPU
          dim3 threadsPerBlock(32, 32); // 1024 is typically the max allowable threads per block
          dim3 numBlocks((workingShape[0] / threadsPerBlock.x) + 1,
                         (workingShape[1] / threadsPerBlock.y) + 1);
          matmul<<<numBlocks, threadsPerBlock>>>( workingShape[0], workingShape[1], currentShape[1], newShape[1],
                                                  d_currentMatrix, d_newMatrix, d_workingMatrix);

          // Copy Solution back from GPU
          hipMemcpy(workingMatrix, d_workingMatrix, workingLength, hipMemcpyDeviceToHost);

          // Free Memory on GPU
          hipFree(d_workingMatrix);
          hipFree(d_currentMatrix);
          hipFree(    d_newMatrix);
        } else {
          // Naive CPU Fallback for Small Matrices
          for (int i = 0; i < currentShape[0]; i++) {       // i is the row in this matrix
            for (int j = 0; j <     newShape[1]; j++) {     // j is the column in the other matrix
              workingMatrix[(i * workingShape[1]) + j] = 0; // malloc doesn't initialize to zeros on Linux...
              for (int k = 0; k < currentShape[1]; k++) {   // k is the column in this matrix
                workingMatrix[(i * workingShape[1]) + j] +=
                currentMatrix[(i * currentShape[1]) + k] *
                    newMatrix[(k *     newShape[1]) + j];
              }
            }
          }
        }
        free(newMatrix);
      }

      free(currentMatrix);
      currentMatrix   = readyToMultiply ? workingMatrix   : newMatrix;
      currentShape[0] = readyToMultiply ? workingShape[0] : newShape[0];
      currentShape[1] = readyToMultiply ? workingShape[1] : newShape[1];
    }

    fclose (input); // Reached the end of the input file; close the input here

    // Write the output matrix here
    FILE* output = fopen("output.bin", "wb");
    fwrite(currentShape, 4, 2, output);
    fwrite(currentMatrix, 8, currentShape[0] * currentShape[1], output);
    fclose(output);

    free(currentMatrix);
    free(currentShape);
    free(workingShape);
    free(    newShape);
  }
}
